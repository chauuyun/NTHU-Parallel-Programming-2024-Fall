#include <iostream>
#include <algorithm>
#include <fstream>
#include <climits>
#include <hip/hip_runtime.h>

#define NT 1024
#define MAX_N 10010
using namespace std;
/* 1547. Minimum Cost to Cut a Stick */
/* Input: N = 7, DATA = [1,3,4,5]
 * Output: 16
 */

// CPU
// N = DATA 範圍
// C = DATA 的數目
int N, C;
int DATA[MAX_N];
void input(const char *file)
{
    FILE *f = fopen(file, "rb");
    fread(&N, sizeof(int), 1, f);
    fread(&C, sizeof(int), 1, f);
    for (int i = 0; i < C; ++i)
    {
        fread(&DATA[i], sizeof(int), 1, f);
    }
    fclose(f);
}
void output(const char *file, int ans)
{
    FILE *f = fopen(file, "wb");
    fwrite(&ans, sizeof(int), 1, f);
    fclose(f);
}
int minCost(int n, int c, int *cuts);
// GPU
__constant__ int DATA_GPU[MAX_N];
int *dp;
__global__ void min_reduce(int c, int len, int *dp);

int main(int argc, char *argv[])
{
    const char *input_filename = argv[1];
    const char *output_filename = argv[2];

    input(input_filename);
    int res = minCost(N, C, DATA);
    output(output_filename, res);

    return 0;
}

int minCost(int n, int c, int *cuts)
{
    cuts[c++] = 0;
    cuts[c++] = n;
    sort(cuts, cuts + c);
    int dp_size = (c + 1) * c;
    int res;

    // Copy DATA to constant memory on GPU
    hipMemcpyToSymbol(HIP_SYMBOL(DATA_GPU), cuts, c * sizeof(int));
    // Allocate and initialize dp array on GPU
    hipMalloc(&dp, dp_size * sizeof(int));
    hipMemset(dp, 0, dp_size * sizeof(int));
    // Dynamic programming using CUDA
    for (int len = 3; len <= c; ++len)
    {
        int num = c - len + 1;
        int block_num = (num + NT - 1) / NT;
        min_reduce<<<block_num, NT>>>(c, len, dp);
    }
    // Retrieve the result
    // get dp[c][0]
    hipMemcpy(&res, &dp[c * c + 0], sizeof(int), hipMemcpyDeviceToHost);
    return res;
}

__global__ void min_reduce(int c, int len, int *dp)
{
    int l = blockIdx.x * blockDim.x + threadIdx.x;
    int r = l + len - 1;
    if (r >= c)
        return;
    dp[len * c + l] = INT_MAX;
    for (int leftLen = 2; leftLen < len; ++leftLen)
    {
        int rightLen = len - leftLen + 1, rightIdx = l + leftLen - 1;
        dp[len * c + l] = min(
            dp[len * c + l],
            dp[leftLen * c + l] + dp[rightLen * c + rightIdx] + DATA_GPU[r] - DATA_GPU[l]);
    }
}